#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

struct point { double x,y; };

__global__ void MyFunc(point* d_a){
     if(threadIdx.x == 0 && threadIdx.y == 0)
     {
        d_a->x=100.0;
        d_a->y = 100.0;
     }
}
int main(void){
   point * a = (point*)malloc(sizeof(point));
   a->x=10.0;
   a->y=10.0;
   point * d_a;
   hipMalloc((void**)&d_a,sizeof(point));
   hipMemcpy(d_a,a,sizeof(point),hipMemcpyHostToDevice);
   dim3 dimblock(16,16);
   dim3 dimgrid(1,1);
   MyFunc<<<dimgrid,dimblock>>>(d_a);
   hipMemcpy(a,d_a,sizeof(point),hipMemcpyDeviceToHost);
   printf("%lf %lf\n",a->x,a->y);
   return hipDeviceReset();
}
