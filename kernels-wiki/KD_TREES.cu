#include "hip/hip_runtime.h"
float distanza(float4 a, float4 b)
{
	float dx, dy, dz;
 
	dx = a.x-b.x;
	dx *= dx;
 
	dy = a.y-b.y;
	dy *= dy;
 
	dz = a.z-b.z;
	dz *= dz;
 
	return dx+dy+dz;
}
 
bool isNull(float4 p)
{
	if(p.w==-1)
		return true;
 
	return false;
}
 
int findLeaf(__global float4 *model, float4 qPoint, int model_size, int cur)
{
	int best_id = -1;
	int asse;
 
	while(cur <= model_size)
	{		
		asse = model[cur].w;
 
		if(qPoint[asse] < model[cur][asse])
		{			
			if(cur*2+1>=model_size || isNull(model[cur*2+1]))
			{
				if(cur*2+2>=model_size || isNull(model[cur*2+2]))
				{
					best_id = cur;
					break;
				}
				else
				{
					cur = cur*2+2;
				}
			}
			else
			{
				cur = cur*2+1;
			}
		}
		else
		{
			if(cur*2+2>=model_size || isNull(model[cur*2+2]))
			{
				if(cur*2+1>=model_size || isNull(model[cur*2+1]))
				{
					best_id = cur;
					break;
				}
				else
				{
					cur = cur*2+1;
				}
			}
			else
			{
				cur = cur*2+2;
			}
		}
	}
 
	return best_id;
}
 
__kernel void 
nearest_neighbour(__global float4 *model,
	__global float4 *dataset,
	__global int *nearest,
	const int model_size)
{
		int g_dataset_id = get_global_id(0);
 
		float4 qPoint = dataset[g_dataset_id];
 
		int stack[7]; // 7 is enough for the number of points in my model
		int top = 0;
 
		stack[top] = -1;
 
		int node = findLeaf(model, qPoint, model_size, 0);
 
		int nn = node;
 
		int lastChild, asse, otherSide;
 
		float bestDist = distanza(qPoint, model[node]);
 
		while(node != 0)
		{
			lastChild = node;
 
			node = (node - 1) / 2;
 
			if(stack[top] == node)
			{
				--top;
			}
			else
			{
				float parentDist = distanza(qPoint, model[node]);
 
				if(parentDist < bestDist)
				{
					bestDist = parentDist;
					nn = node;
				}
 
				asse = model[node].w;
 
				float testDist = model[node][asse] - qPoint[asse];
				testDist = testDist * testDist;
 
				if(testDist < bestDist)
				{
					if (node*2+1 == lastChild)
					{
						otherSide = node*2+2;
					}
					else
					{
						otherSide = node*2+1;
					}
 
					if(otherSide < model_size && !isNull(model[otherSide]))
					{
						int testNode = findLeaf(model, qPoint, model_size, otherSide);
 
						testDist = distanza(qPoint, model[testNode]);
 
						if(testDist < bestDist)
						{
							bestDist = testDist;
							nn = testNode;
						}
 
						++top;
						stack[top] = node;
 
						node = testNode;
					}
				}
			}
		}
 
		nearest[g_dataset_id] = nn;
}