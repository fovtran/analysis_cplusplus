// %comspec% /k ""C:\Program Files (x86)\Microsoft Visual Studio 14.0\VC\vcvarsall.bat"" amd64
// nvcc -arch=compute_20 -code=sm_20 cublas_trans.cu -o cublas_trans.exe -lcublas
// cp -r c:\VS2013\vc\bin\x86_amd64 c:\VS2013\VC\bin\amd64
// rename c:\VS2013\VC\bin\amd64\vcvarsx86_amd64.bat c:\VS2013\vc\bin\amd64\vcvars64.bat
// nvcc x.cu ...   -ccbin "D:\Program Files\Microsoft Visual Studio 11.0\VC\bin"
#include <hipblas.h>

//#include <stdio.h>
//#include <stdlib.h>
//#include <math.h>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main(void)
{
	int x_rows_num = 4;
	int q_rows_num = 2;
	int dim = 3;

	int N = x_rows_num*dim;
	int M = q_rows_num*dim;

	float x0[12] = {
			0, 1, 2,
			3, 4, 5,
			6, 7, 8,
			9, 10, 11};

	float q0[6]  = {
			3, 4,
			5, 6,
			7, 8 };

	float *x, *q, *x_q_multiplication;
	CHECK( hipMallocManaged(&x, N*sizeof(float)) );
	CHECK( hipMallocManaged(&q, M*sizeof(float)) );
	CHECK( hipMallocManaged(&x_q_multiplication, q_rows_num*x_rows_num*dim) );

	std::memcpy(x, x0,  N*sizeof(float));
	std::memcpy(q, q0,  M*sizeof(float));

	float *q_device;
	hipMallocManaged(&q_device, M*sizeof(float));
	hipMemcpy(q_device, q, M*sizeof(float), hipMemcpyHostToDevice);

	hipblasHandle_t handle; hipblasCreate(&handle);

	float alpha = 1.f; float beta = 0.f;

	hipblasSgemm(handle,
	        HIPBLAS_OP_N, HIPBLAS_OP_T,
	        q_rows_num, x_rows_num, dim,
	        &alpha, // 1
	        q_device, q_rows_num,
	        x, x_rows_num,
	        &beta, // 0
	        x_q_multiplication, q_rows_num);

	hipDeviceSynchronize();

	for (int i = 0; i < q_rows_num*x_rows_num; i++) std::cout << x_q_multiplication[i] << " "; std::cout << std::endl;
	// for (int i = 0; i < q_rows_num*x_rows_num; i++) printf("%d: %f", x_q_multiplication[i]);

	hipFree(x); hipFree(q); hipFree(x_q_multiplication); return 0;
}
